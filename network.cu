#include "hip/hip_runtime.h"
/*
@author Jan Nemec, jnem6403@seznam.cz

*/


#include <stdio.h>
#include <stdlib.h>

/*
	Global settings
*/

/** Compile as emulation or use CUDA */
#define EMULATION 0

/** Number of non input and non output groups of neuron */
#define HIDDEN_GROUPS 5

#define GROUP_COUNT (HIDDEN_GROUPS + 2)

/** Number of neuron in each group */

#define NEURONS_IN_GROUP 100

/** Divide each float coef by this */
#define DIVIDE_COEF 8192

/** bigger TRESHOLD_RAND -> bigger tresholds */
#define TRESHOLD_RAND 32768

/** maximal number of external connections */
#define MAX_EXTERNAL_CONNECTIONS 8

/** bigger WEIGHT_RAND -> bigger weights */
#define WEIGHT_RAND 110

/** bigger INPUT_RAND -> bigger input in the input layer */
#define INPUT_RAND 256

/** how many steps to copmpute */
#define ITERATIONS 1000

/*
	Global types
*/

/** we will compute in this type */
typedef float FLOAT_TYPE;

/** Network of neurons */
typedef struct
{
	/* full matrix NEURONS_IN_GROUP * NEURONS_IN_GROUP
	   weight from 1 to 2 is in w[group][1 + 2 * NEURONS_IN_GROUP]  */
	FLOAT_TYPE w[GROUP_COUNT * NEURONS_IN_GROUP * NEURONS_IN_GROUP];
	/*	0 .. NEURONS_IN_GROUP
		Fixed input (addes every step to potential of the neuron) */
	FLOAT_TYPE inputs[GROUP_COUNT * NEURONS_IN_GROUP];
	/* 0 .. NEURONS_IN_GROUP */
	FLOAT_TYPE tresholds[GROUP_COUNT * NEURONS_IN_GROUP];
	/* 0 .. NEURONS_IN_GROUP */
	FLOAT_TYPE potentials[GROUP_COUNT * NEURONS_IN_GROUP];
	/* is each neuron active in the curren step */
	unsigned char active[GROUP_COUNT * NEURONS_IN_GROUP]; 
	/** Connections from another group
	   connections_xx[group][1][2] is the third (0, 1, 2) connection of the second (0, 1)
       neuron */
	int connection_group[GROUP_COUNT * NEURONS_IN_GROUP * MAX_EXTERNAL_CONNECTIONS];
	int connection_neuron[GROUP_COUNT * NEURONS_IN_GROUP * MAX_EXTERNAL_CONNECTIONS];
	FLOAT_TYPE connection_w[GROUP_COUNT * NEURONS_IN_GROUP * MAX_EXTERNAL_CONNECTIONS];
	/** number of external connections */
	int connectionCount[GROUP_COUNT * NEURONS_IN_GROUP];
} TNetwork;

/**
 Inits every single group of the network. 
 */
void initNetwork(TNetwork *net)
{
	int group;

	for (group = 0; group < GROUP_COUNT; group++)
	{
		int i;

		for (i = 0; i < NEURONS_IN_GROUP; i++)
		{
			int j;

			/* init connections from other groups */
			int limit = net->connectionCount[group * NEURONS_IN_GROUP + i] = rand() % MAX_EXTERNAL_CONNECTIONS;
			for (j = 0; j < limit; j++)
			{
				int index = group * NEURONS_IN_GROUP * MAX_EXTERNAL_CONNECTIONS
					+ i * MAX_EXTERNAL_CONNECTIONS + j;
				
				net->connection_group[index] = rand() % GROUP_COUNT;
				net->connection_neuron[index] = rand() % NEURONS_IN_GROUP;
				net->connection_w[index] = ((rand() % WEIGHT_RAND) / (FLOAT_TYPE) DIVIDE_COEF);
			}
		}
		/* init connections inside this group */
	
	 	for (i = 0; i < NEURONS_IN_GROUP * NEURONS_IN_GROUP; i++)
		{
			net->w[group * NEURONS_IN_GROUP * NEURONS_IN_GROUP + i] =
				(rand() % WEIGHT_RAND) / (FLOAT_TYPE) DIVIDE_COEF;
		}
		/* init all the data for each neuron */
	
		for (i = 0; i < NEURONS_IN_GROUP; i++)
		{
			int index = group * NEURONS_IN_GROUP + i;
			net->inputs[index] = group ? 0 :
				/* "normal" distribution to get more stable result */
				(
					(rand()  % INPUT_RAND) + (rand()  % INPUT_RAND) + 
					(rand()  % INPUT_RAND) + (rand()  % INPUT_RAND)
				) /	(FLOAT_TYPE) (DIVIDE_COEF * 4);
			net->tresholds[index] = (1 + (rand()  % TRESHOLD_RAND)) /
				(FLOAT_TYPE) DIVIDE_COEF;
			net->potentials[index] = 0;
			net->active[index] = 0;
		}
	}
}

/* print the sinle line of the output */
void printOutputArray(int line, const unsigned char *output)
{
	int i;

	printf("%i ", line);
	for (i = 0; i < NEURONS_IN_GROUP; i++)
	{
		putchar(output[i] ? '1' : '0');
	}
	puts("");
}

#if EMULATION

/**
 Single step of the computing
 */
void step(TNetwork *net)
{
	int i;

	/* The first step - connections from other group */

	/* for each group */
	for (i = 0; i < GROUP_COUNT; i++)
	{
		int j;

		/* for each neuron in the group */
		for (j = 0; j < NEURONS_IN_GROUP; j++)
		{
			int k;
			int limit = net->connectionCount[i * NEURONS_IN_GROUP + j];
			/* for each connection (from the other group) of the neuron */
			for (k = 0; k < limit; k++)
			{
				int index = i * NEURONS_IN_GROUP * MAX_EXTERNAL_CONNECTIONS
					+ j * MAX_EXTERNAL_CONNECTIONS + k;
				/* if the other neuron is active*/
				if (
				    net->active
				    	[ net->connection_group[index] * NEURONS_IN_GROUP + 
						  net->connection_neuron[index] ]
				    )
				{
					/* add a bonus to our potential */
					net->potentials[i * NEURONS_IN_GROUP + j] +=
						net->connection_w[index];
				}
			}
		}
	}

	/* The second step */

	/* for each group */
	for (i = 0; i < GROUP_COUNT; i++)
	{
		int j, k;

		/* for each neuron in the group */
		for (j = 0; j < NEURONS_IN_GROUP; j++)
		{

			FLOAT_TYPE *ptrW = net->w + 
				i * (NEURONS_IN_GROUP * NEURONS_IN_GROUP) +
				j * NEURONS_IN_GROUP;
			
			unsigned char *ptrA = net->active + i * NEURONS_IN_GROUP;
			int index = i * NEURONS_IN_GROUP + j;

			/* for each connection */
			for (k = 0; k < NEURONS_IN_GROUP; k++)
			{
				if (*ptrA)
				{
					/* add the weight if the neuron is active */
					net->potentials[index] += *ptrW;
				} 
				ptrW++;
				ptrA++;
			}
			/* Add input to the potential */ 
			net->potentials[index] += net->inputs[index];
		}
	}

	/* for each group */
	for (i = 0; i < GROUP_COUNT; i++)
	{
		int j;

		/* for each neuron in the group */
		for (j = 0; j < NEURONS_IN_GROUP; j++)
		{
			int index = i * NEURONS_IN_GROUP + j;
		/* Check tresholds and set active neuron*/
			if (net->potentials[index] >= net->tresholds[index])
			{
				net->potentials[index] = 0;
				net->active[index] = 1;
			}
			else
			{
				net->active[index] = 0;
			}
		}
	}
}

/* print the output of the network */
void printResult(int line, TNetwork *net)
{
	printOutputArray(line, net->active + (GROUP_COUNT - 1) * NEURONS_IN_GROUP);
}

#else

/**
	One step of computing - updating of potentials
*/
__global__ void updatePotentials(TNetwork *net)
{
	int g = blockIdx.x;
	int n = threadIdx.x;
	
	int k;
	int index = NEURONS_IN_GROUP * g + n;
	int limit = net->connectionCount[index];
	
	
	/* for each connection (from the other group) of the neuron */
	for (k = 0; k < limit; k++)
	{
		int index2 = g * NEURONS_IN_GROUP * MAX_EXTERNAL_CONNECTIONS
					+ n * MAX_EXTERNAL_CONNECTIONS + k;
		if (
		    net->active
		    	[NEURONS_IN_GROUP * net->connection_group[index2] +
				 net->connection_neuron[index2] ]
		    )
		{
			/* add a bonus to our potential */
			net->potentials[index] += net->connection_w[index2];
		}
	}

	FLOAT_TYPE *ptrW = net->w + 
		g * (NEURONS_IN_GROUP * NEURONS_IN_GROUP) +
		n * NEURONS_IN_GROUP;
			
	unsigned char *ptrA = net->active + g * NEURONS_IN_GROUP;

	/* for each connection */
	for (k = 0; k < NEURONS_IN_GROUP; k++)
	{
		if (*ptrA)
		{
			/* add the weight if the neuron is active */
			net->potentials[index] += *ptrW;
		} 
		ptrW++;
		ptrA++;
	}
	/* Add input to the potential */ 
	net->potentials[index] += net->inputs[index];
}

/**
	One step of computing - updating of active states
*/
__global__ void updateActive(TNetwork *net)
{
	int g = blockIdx.x;
	int n = threadIdx.x;
	int index = NEURONS_IN_GROUP * g + n;
 
	if (net->potentials[index] >= net->tresholds[index])
	{
		net->potentials[index] = 0;
		net->active[index] = 1;
	}
	else
	{
		net->active[index] = 0;
	}
}

/**
	Copy active states from the output group from the device memory
 	TODO - make this faster 
*/
__global__ void getOutput(TNetwork *net, unsigned char *output)
{
	int n = threadIdx.x;

	output[n] = net->active[(GROUP_COUNT - 1) * NEURONS_IN_GROUP + n];
}

/** report error and exit */
void handleError(hipError_t e, const char *function)
{
	fprintf(stderr, "Error %u in %s (%s), exiting\n",
		(unsigned) e, function, hipGetErrorString(e));
	exit(1);
}

/** check hipGetLastError() */
void checkAndHandleKernelError(const char *function)
{
	hipError_t e;
	e = hipGetLastError();
	if (e != hipSuccess)
	{
		handleError(e, function);
	}
}

/** check the function call return code */
void checkAndHandleFunctionError(hipError_t e, const char *function)
{
	if (e != hipSuccess)
	{
		handleError(e, function);
	}
}

#endif


int main(void)
{
	int i;
	TNetwork *net = (TNetwork *)malloc(sizeof(TNetwork));
	srand(time(NULL));
	initNetwork(net);

#if EMULATION
	for (i = 0; i < ITERATIONS; i++)
	{
		step(net);
		printResult(i, net);
	}
#else
	
	TNetwork *d_net;

	checkAndHandleFunctionError(hipMalloc(&d_net, sizeof(TNetwork)),
		"hipMalloc");
	checkAndHandleFunctionError(hipMemcpy(d_net, net, sizeof(TNetwork),
		hipMemcpyHostToDevice), "hipMemcpy"); 
	for (i = 0; i < ITERATIONS; i++)
	{
		unsigned char active[NEURONS_IN_GROUP];


		updatePotentials<<<GROUP_COUNT, NEURONS_IN_GROUP>>>(d_net);
		checkAndHandleKernelError("updatePotentials");
		
		updateActive<<<GROUP_COUNT, NEURONS_IN_GROUP>>>(d_net);
		checkAndHandleKernelError("updateActive");

		getOutput<<<1, NEURONS_IN_GROUP>>>(d_net, active);
		checkAndHandleKernelError("getOutput");

		printOutputArray(i, active);
	}
	checkAndHandleFunctionError(hipFree(d_net), "hipFree");
#endif

	free(net);
	return 0;
}
