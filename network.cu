#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

/*
	Global settings
*/

int HIDDEN_GROUPS = 5;
int NEURONS_IN_GROUP = 100;
int EMULATION = 1;

/*
	Global types
*/

typedef float FLOAT_TYPE;

/*
	Data inside of each group (without connections to other groups)
*/
typedef struct
{
	/* full matrix NEURONS_IN_GROUP * NEURONS_IN_GROUP
	   weight from 1 to 2 is in w[1 + 2 * NEURONS_IN_GROUP]  */
	FLOAT_TYPE *w;
	/*	0 .. NEURONS_IN_GROUP
		Fixed input (addes every step to potential of the neuron) */
	FLOAT_TYPE *inputs;
	/* 0 .. NEURONS_IN_GROUP */
	FLOAT_TYPE *tresholds;
	/* 0 .. NEURONS_IN_GROUP */
	FLOAT_TYPE *potentials;
	/* is each neuron active in the curren step */
	unsigned char *active; 
} TGroupInternal;

/* Connection between groups*/
typedef struct
{
	/* from group index */
	int group;
	/* from neuron (inside of the group) index */
	int neuron;
	/* weight */
	FLOAT_TYPE w;
} TConnection;

/* Group inluding without connections to other groups */
typedef struct
{
	/* Internal group data */
	TGroupInternal inside;
	/* Connections from another group
	   connections[1][2] is the third (0, 1, 2) connection of the second (0, 1)
       neuron */
	TConnection **connections;
	int *connectionCount;
} TGroup;

typedef struct
{
	/* All groups, the first is input, the last is output. */
	TGroup *groups;
	/* Including input and output */
	int groupCount;
} TNetwork;


// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

void initGroup(int hiddenGroups, int neuronsInGroup, int index, TGroup *group)
{
	int i;

	group->connections = (TConnection **) malloc(sizeof(TConnection *) * 
		neuronsInGroup);
	group->connectionCount = (int *) malloc(sizeof(int) * neuronsInGroup);
	for (i = 0; i < neuronsInGroup; i++)
	{
		int j;

		/* init connections from other groups */
		group->connectionCount[i] = rand() % 8;
		group->connections[i] = (TConnection *)
			malloc(sizeof(TConnection) * group->connectionCount[i]);
		for (j = 0; j < group->connectionCount[i]; j++)
		{
			TConnection *conn = group->connections[i] + j;
			conn->group = rand() % (hiddenGroups + 2);
			conn->neuron = rand() % neuronsInGroup;
			conn->w = (rand() & 0xFF) / (FLOAT_TYPE)512.0;
		}
	}
	/* init connections inside this group */
	group->inside.w = (FLOAT_TYPE *) malloc(sizeof(FLOAT_TYPE) *
		neuronsInGroup * neuronsInGroup);
 	for (i = 0; i < neuronsInGroup * neuronsInGroup; i++)
	{
		group->inside.w[i] = (rand() & 0xFF) / (FLOAT_TYPE)512.0;
	}
	/* init all the data for each neuron */
	group->inside.inputs = (FLOAT_TYPE *) malloc(sizeof(FLOAT_TYPE) *
		neuronsInGroup);
	group->inside.tresholds = (FLOAT_TYPE *) malloc(sizeof(FLOAT_TYPE) *
		neuronsInGroup);
	group->inside.potentials = (FLOAT_TYPE *) malloc(sizeof(FLOAT_TYPE) *
		neuronsInGroup);
	group->inside.active = (unsigned char *) malloc(neuronsInGroup);
	for (i = 0; i < neuronsInGroup; i++)
	{
		group->inside.inputs[i] = (i ? 0 : (rand() & 1));
		group->inside.tresholds[i] = (rand() & 0xFF) / (FLOAT_TYPE) 128.0;
		group->inside.potentials[i] = 0;
		group->inside.active[i] = 0;
	}
}

void doneGroup(int neuronsInGroup, TGroup *group)
{
	int i;
	for (i = 0; i < neuronsInGroup; i++)
	{
		free(group->connections[i]);
	}
	free(group->connections);
	free(group->connectionCount);
	free(group->inside.inputs);
	free(group->inside.tresholds);
	free(group->inside.potentials);
	free(group->inside.active);
}

void initNetwork(int hiddenGroups, int neuronsInGroup, TNetwork *net)
{
	int i;
	int limit = hiddenGroups + 2;
	net->groupCount = limit;
	net->groups = (TGroup *) malloc(sizeof(TGroup) * limit);
	for (i = 0; i < limit; i++)
	{
		initGroup(hiddenGroups, neuronsInGroup, i, net->groups + i);
	}
}

void doneNetwork(int neuronsInGroup, TNetwork *net)
{
	int i;
	for (i = 0; i < net->groupCount; i++)
	{
		doneGroup(neuronsInGroup, net->groups + i);
	}
	free(net->groups);
}

void step(TNetwork *net)
{
	int i;

	/* The first step which is hard to make paralell 
		- connections from other group, it is hard to separate the memory */

	for (i = 0; i < net->groupCount; i++)
	{
		int j;

		TGroup *group = net->groups + i;
		for (j = 0; j < NEURONS_IN_GROUP; j++)
		{
			int k;
			for (k = 0; k < group->connectionCount[j]; k++)
			{
				TConnection *conn = group->connections[j] + k;
				if (net->groups[conn->group].inside.active[conn->neuron])
				{
					group->inside.potentials[j] += conn->w;
				}
			}
		}
	}

	/* The second step should be done paralell */
	for (i = 0; i < net->groupCount; i++)
	{
		int j, k;
		TGroup *group = net->groups + i;
	
		for (j = 0; j < NEURONS_IN_GROUP; j++)
		{
			/* Add bonus from connection inside of the group to potential of each
		   	   neuron. */

			FLOAT_TYPE *ptrW = group->inside.w + j * NEURONS_IN_GROUP;
			unsigned char *ptrA = group->inside.active;
			for (k = 0; k < NEURONS_IN_GROUP; k++)
			{
				if (*ptrA)
				{
					group->inside.potentials[j] += *ptrW;
				} 
				ptrW++;
				ptrA++;
			}
			/* Add input to the potential */ 
			group->inside.potentials[j] += group->inside.inputs[j];

			/* Check tresholds and set active neuron*/
			if (group->inside.potentials[j] >= group->inside.tresholds[j])
			{
				group->inside.potentials[j] = 0;
				group->inside.active[j] = 1;
			}
			else
			{
				group->inside.active[j] = 0;
			}
		}
	}
}

void printResult(TNetwork *net)
{
	int i;
	TGroup *last = net->groups + (net->groupCount - 1);
	for (i = 0; i < NEURONS_IN_GROUP; i++)
	{
		putchar(last->inside.active[i] ? '1' : '0');
	}
	puts("");
}


int main(void)
{
	int i;
	TNetwork net;
	srand(time(NULL));
	initNetwork(HIDDEN_GROUPS, NEURONS_IN_GROUP, &net);
	for (i = 0; i < 10; i++)
	{
		step(&net);
		printResult(&net);
	}
	doneNetwork(NEURONS_IN_GROUP, &net);
	/* VecAdd<<<1, N>>>(A, B, C); */

	return 0;
}
